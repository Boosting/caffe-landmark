#include "hip/hip_runtime.h"
#include "caffe/deep_landmark_layers.hpp"

namespace caffe {
  
template <typename Dtype>
__global__ void ForwardSqrt(int n, const Dtype* in, Dtype* out) {
  // f(x) = sqrt(x)
  CUDA_KERNEL_LOOP(index, n){
    Dtype val = in[index];
    out[index] = sqrt(val);
  }
}

template <typename Dtype>
void L2DistanceLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                                        const vector<Blob<Dtype>*>& top) {
  // bottom[0] stores ground truth of the shape (batch, num_landmark, 2, 1)
  // bottom[1] stores predictions of the shape (batch, num_landmark, 2, 1)
  // diff_x_ = x - x'
  // diff_y_ = y - y'
  for (int b = 0; b < bottom[0]->num(); ++b){    
    for (int n = 0; n < num_landmark_; ++n){
      diff_x_.mutable_cpu_data()[diff_x_.offset(b) + n] = 
          bottom[0]->cpu_data()[bottom[0]->offset(b, n)] - 
          bottom[1]->cpu_data()[bottom[1]->offset(b, n)];
      
      diff_y_.mutable_cpu_data()[diff_y_.offset(b) + n] = 
          bottom[0]->cpu_data()[bottom[0]->offset(b, n, 1)] -
          bottom[1]->cpu_data()[bottom[1]->offset(b, n, 1)];
    }
  }
  Blob<Dtype> diff_x2, diff_y2;
  diff_x2.Reshape(diff_x_.shape());
  diff_y2.Reshape(diff_x_.shape());
  // diff_x2 = (x - x')^2
  // diff_y2 = (y - y')^2
  int count = diff_x_.count();
  caffe_gpu_mul(
    count,
    diff_x_.gpu_data(),
    diff_x_.gpu_data(),
    diff_x2.mutable_gpu_data());

  caffe_gpu_mul(
    count,
    diff_y_.gpu_data(),
    diff_y_.gpu_data(),
    diff_y2.mutable_gpu_data());
  
  // sum_ = (x - x')^2 + (y - y')^2
  caffe_gpu_add(
    count,
    diff_x2.gpu_data(),
    diff_y2.gpu_data(),
    sum_.mutable_gpu_data());
  
  // sum_ = sqrt((x - x')^2 + (y - y')^2)
  ForwardSqrt<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
    count, sum_.gpu_data(), sum_.mutable_gpu_data());
  CUDA_POST_KERNEL_CHECK;
  
  Dtype sum_error = Dtype(0);

  for (int n = 0; n < num_landmark_; ++n) {
    for (int b = 0; b < sum_.num(); ++b) {
      sum_error += sum_.cpu_data()[sum_.offset(b, n)];
    }
    
  }
  top[0]->mutable_cpu_data()[0] = sum_error / sum_.num();
}

template <typename Dtype>
void L2DistanceLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                                         const vector<bool>& propagate_down,
                                                         const vector<Blob<Dtype>*>& bottom) {
   // backward
   Blob<Dtype> deriative_x,deriative_y, diff;
   deriative_x.Reshape(diff_x_.shape());
   deriative_y.Reshape(diff_x_.shape());
   diff.Reshape(bottom[0]->shape());
   int count = diff_x_.count();
   caffe_gpu_div(
      count,
      diff_x_.gpu_data(),
      sum_.gpu_data(),
      deriative_x.mutable_gpu_data());
   caffe_gpu_div(
      count,
      diff_y_.gpu_data(),
      sum_.gpu_data(),
      deriative_y.mutable_gpu_data());
   for(int b=0;b<bottom[0]->num();b++){
      for(int n=0;n<num_landmark_;n++){
         diff.mutable_cpu_data()[diff.offset(b,n)] = deriative_x.cpu_data()[deriative_x.offset(b,n)];
         diff.mutable_cpu_data()[diff.offset(b,n,1)] = deriative_y.cpu_data()[deriative_y.offset(b,n)];
      }
   }
   int i;
   for(i=0;i < 2;i++){
     int sign = (i == 0) ? 1 : -1;
     const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
     caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                              // alpha
          diff.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
   }

   
}



INSTANTIATE_LAYER_GPU_FUNCS(L2DistanceLossLayer);
} // namespace caffe
