#include "hip/hip_runtime.h"
#include "caffe/deep_landmark_layers.hpp"

namespace caffe {
  
template <typename Dtype>
__global__ void ForwardSqrt(int n, const Dtype* in, Dtype* out) {
  // f(x) = sqrt(x)
  CUDA_KERNEL_LOOP(index, n){
    Dtype val = in[index];
    out[index] = sqrt(val);
  }
}

template <typename Dtype>
void L2DistanceLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                                        const vector<Blob<Dtype>*>& top) {
  // bottom[0] stores ground truth of the shape (batch, num_landmark, 2, 1)
  // bottom[1] stores predictions of the shape (batch, num_landmark, 2, 1)
  // diff_x = x - x'
  // diff_y = y - y'
  for (int b = 0; b < bottom[0]->num(); ++b){    
    for (int n = 0; n < num_landmark_; ++n){
      diff_x.mutable_cpu_data()[diff_x.offset(b) + n] = 
          bottom[0]->cpu_data()[bottom[0]->offset(b, n)] - 
          bottom[1]->cpu_data()[bottom[1]->offset(b, n)];
      
      diff_y.mutable_cpu_data()[diff_y.offset(b) + n] = 
          bottom[0]->cpu_data()[bottom[0]->offset(b, n, 1)] -
          bottom[1]->cpu_data()[bottom[1]->offset(b, n, 1)];
    }
  }
  Blob<Dtype> diff_x2, diff_y2;
  diff_x2.Reshape(diff_x.shape());
  diff_y2.Reshape(diff_x.shape());
  // diff_x = (x - x')^2
  // diff_y = (y - y')^2
  int count = diff_x.count();
  caffe_gpu_mul(
    count,
    diff_x.gpu_data(),
    diff_x.gpu_data(),
    diff_x2.mutable_gpu_data());

  caffe_gpu_mul(
    count,
    diff_y.gpu_data(),
    diff_y.gpu_data(),
    diff_y2.mutable_gpu_data());
  
  // sum = (x - x')^2 + (y - y')^2
  caffe_gpu_add(
    count,
    diff_x2.gpu_data(),
    diff_y2.gpu_data(),
    sum.mutable_gpu_data());
  
  // sum = sqrt((x - x')^2 + (y - y')^2)
  ForwardSqrt<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
    count, sum.gpu_data(), sum.mutable_gpu_data());
  CUDA_POST_KERNEL_CHECK;
  
  // bottom[2] stores bi-ocular distance of the shape (batch, 1, 1, 1);
  // sum = sqrt((x - x')^2 + (y - y')^2) / bi-ocular-distance
  /*
  for (int b = 0; b < bottom[2]->num(); ++b) {
    for (int n = 0; n < num_landmark_; ++n) {
      sum.mutable_cpu_data()[sum.offset(b) + n] = 
          sum.cpu_data()[sum.offset(b) + n] / bottom[2]->cpu_data()[bottom[2]->offset(b)];
      
    }
  }*/

  Dtype sum_error = Dtype(0);
  // calculate average error of each landmark
  // top[0] stores average error for each landmark
  for (int n = 0; n < num_landmark_; ++n) {
    for (int b = 0; b < sum.num(); ++b) {
      sum_error += sum.cpu_data()[sum.offset(b, n)];
    }
    
    //top[0]->mutable_cpu_data()[top[0]->offset(n)] = sum_error / sum.num();
  }
  top[0]->mutable_cpu_data()[0] = sum_error / sum.num();
}

template <typename Dtype>
void L2DistanceLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                                         const vector<bool>& propagate_down,
                                                         const vector<Blob<Dtype>*>& bottom) {
   // backward
   Blob<Dtype> deriative_x,deriative_y, diff;
   deriative_x.Reshape(diff_x.shape());
   deriative_y.Reshape(diff_x.shape());
   diff.Reshape(bottom[0]->shape());
   int count = diff_x.count();
   caffe_gpu_div(
      count,
      diff_x.gpu_data(),
      sum.gpu_data(),
      deriative_x.mutable_gpu_data());
   caffe_gpu_div(
      count,
      diff_y.gpu_data(),
      sum.gpu_data(),
      deriative_y.mutable_gpu_data());
   for(int b=0;b<bottom[0]->num();b++){
      for(int n=0;n<num_landmark_;n++){
         diff.mutable_cpu_data()[diff.offset(b,n)] = deriative_x.cpu_data()[deriative_x.offset(b,n)];
         diff.mutable_cpu_data()[diff.offset(b,n,1)] = deriative_y.cpu_data()[deriative_y.offset(b,n)];
      }
   }
   int i;
   for(i=0;i < 2;i++){
     int sign = (i == 0) ? 1 : -1;
     const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
     caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                              // alpha
          diff.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
   }

   
}


template <typename Dtype>
void PrintBlob(const Blob<Dtype> &blob, string name){
  std::cout << "#################################" << std::endl;
  std::cout << "name:" << name << std::endl;
  std::cout << "num:" << blob.num() << std::endl;
  std::cout << "channel:" << blob.channels() << std::endl;
  std::cout << "height:" << blob.height() << std::endl;
  std::cout << "width:" << blob.width() << std::endl;
  std::cout << "data:" << std::endl;
  std::cout << "[";
  for (int n = 0; n < blob.num(); ++n){
    for (int c = 0; c < blob.channels(); ++c){
      for (int h = 0; h < blob.height(); ++h){
        for (int w = 0; w < blob.width(); ++w){
          std::cout << blob.data_at(n,c,h,w) << ",";
        }
      }
    }
  }
  std::cout << "]" << std::endl;
  std::cout << "#################################" << std::endl;
   
}

INSTANTIATE_LAYER_GPU_FUNCS(L2DistanceLossLayer);
} // namespace caffe
